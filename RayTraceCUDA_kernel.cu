#include "hip/hip_runtime.h"
/** \file RayTraceCUDA_kernel.cu
 * \author Tomasz Jakubczyk
 * \brief RayTrace CUDA kernel function & helpers
 */
#define WIN32
#include<stdlib.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "hip/hip_vector_types.h"
#include "hip/hip_math_constants.h"
#include "rcstruct.cuh"
#include "HandlesStructures.cuh"

extern "C"
{
__device__
float3 findAlpha( float3 n, float3 v, float p, float m2 )
{
    float al1=acos(dot(n,v));
    float al2;
    if(p==1)
    {
        al2=asin(sin(al1)/m2);
    }
    else
    {
        al2=asin(m2*sin(al1));
    }
    float bet=al1-al2;
    float3 S=cross(v,n);
    float3 V2;
    if(length(S)==0.0f)
    {
        V2=v;
    }
    else
    {
        float W=S.x*S.x+S.y*S.y+S.z*S.z;
        float2 B=make_float2(cos(bet),cos(al2));
        float Wx=(B.x*n.y-B.y*v.y)*S.z+(B.y*v.z-B.x*n.z)*S.y;
        float Wy=(B.y*v.x-B.x*n.x)*S.z+(B.x*n.z-B.y*v.z)*S.x;
        float Wz=(B.y*v.y-B.x*n.y)*S.x+(B.x*n.x-B.y*v.x)*S.y;
        V2=make_float3(Wx/W,Wy/W,Wz/W);
    }
    return V2;
}

__device__
rcstruct SphereCross( float3 r, float3 V, float R )
{
    float A=V.x*V.x+V.y*V.y+V.z*V.z;
    float B=2.0f*dot(r,V);
    float C=r.x*r.x+r.y*r.y+r.z*r.z-R*R;
    float D=B*B-4.0f*A*C;
    rcstruct rc;
    if(D<0.0f)
    {
        rc.a=make_float3(HIP_NAN_F,HIP_NAN_F,HIP_NAN_F);
        rc.b=make_float3(HIP_NAN_F,HIP_NAN_F,HIP_NAN_F);
    }
    else
    {
        float t1=(-B+sqrt(D))/2.0f/A;
        float t2=(-B-sqrt(D))/2.0f/A;
        rc.a=r+V*t1;
        rc.b=r+V*t2;
    }
    return rc;
}

__global__
/** \brief RayTrace CUDA kernel function.
 *
 * \param Br float*
 * \param Vb float*
 * \param VH float*
 * \param Vb_length int
 * \param VH_length int
 * \param S HandlesStructures structure contains the parameters of the lens
 * \param IC float* correction matrix
 * \param PK float4* pixel position matrix
 * \return void
 *
 */
void RayTraceD(float* Br, float* Vb, float* VH, int Vb_length, int VH_length, HandlesStructures S, float* IC, float4* PK)
{
    uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    //float3 P[11];
    /*if(index==0)
    {
        P[0]=make_float3(-1,-1,-1);//error1
    }*/
    uint indexi = index/Vb_length;
    if (indexi >= VH_length)
    {
        //P[index*7]=make_float3(-100,-100,-100);//error1
        return;//empty kernel
    }
    uint indexj = index%Vb_length;
    if (indexj >= Vb_length)
    {
        //P[index*7]=make_float3(-200,-200,-200);//error1
        return;//critical error
    }

    float3 P2=make_float3(Br[indexj],Vb[indexj],VH[indexi]);/**< point on the surface of the first diaphragm */

    //uint p=0;
    float3 nan3=make_float3(HIP_NAN_F,HIP_NAN_F,HIP_NAN_F);

    //Calculation of the position of the sphere's center
    S.Cs1=S.l1-S.R1+S.g;
    S.Cs2=S.Cs1+S.ll+2.0f*S.R2;

    float3 P1 = S.Pk;//droplet coordinates

    float3 v = normalize(P2 - P1);//direction vector of the line
    //looking for the point of intersection of the line and lenses
    float t = (S.l1 - P2.x)/v.x;
    float3 P3 = P2 + t*v;//Point in the plane parallel to the flat surface of the lens

    if (length(make_float2(P3.y,P3.z)) > (S.efD/2))//verification whether  the point inside the aperture of the lens or not
    {
        //recalculate coordinates
        float Kp = length(make_float2(P3.y,P3.z))/(S.efD/2);
        P3.y/=Kp;
        P3.z/=Kp;
        v = normalize(P3 - P1);//direction vector of the line
    }

    //normal vector to the surface
    float3 n=make_float3(1.0f,0.0f,0.0f);

    float3 v3 = findAlpha( n, v,1,S.m2 );

    //For intensity calculation
    float P8 = acos(dot(n,v));

    rcstruct rc = SphereCross( make_float3( P3.x - S.Cs1, P3.y, P3.z ), v3,S.R1 );

    if(isnan(rc.a.x))
    {
        /*p=0;
        P[index*7+p++]=P1;
        P[index*7+p++]=P2;
        P[index*7+p++]=P3;
        P[index*7+p++]=nan3;
        P[index*7+p++]=nan3;
        P[index*7+p++]=nan3;
        P[index*7+p++]=nan3;*/
        //P[index*7]=make_float3(100,100,100);//error1
        return;
    }

    float3 ns = normalize(rc.a);
    float3 v4 = findAlpha( ns, v3,2,S.m2 );

    //For intensity calculation
    float P9 = acos(dot(ns, v3));

    float3 P4 = make_float3( rc.a.x + S.Cs1, rc.a.y, rc.a.z );

    if(length(make_float2(rc.a.y,rc.a.z)) > S.D/2)
    {
        /*p=0;
        P[index*7+p++]=P1;
        P[index*7+p++]=P2;
        P[index*7+p++]=P3;
        P[index*7+p++]=P4;
        P[index*7+p++]=nan3;
        P[index*7+p++]=nan3;
        P[index*7+p++]=nan3;*/
        //P[index*7]=make_float3(200,200,200);//error2
        return;
    }

    rcstruct rc1 = SphereCross( make_float3(P4.x-S.Cs2,P4.y,P4.z), v4,S.R2 );
    if(isnan( rc1.a.x ))
    {
        /*p=0;
        P[index*7+p++]=P1;
        P[index*7+p++]=P2;
        P[index*7+p++]=P3;
        P[index*7+p++]=P4;
        P[index*7+p++]=nan3;
        P[index*7+p++]=nan3;
        P[index*7+p++]=nan3;*/
        //P[index*7]=make_float3(300,300,300);//error3
        return;
    }
    float3 P5 = rc1.b;
    P5.x = P5.x + S.Cs2;


    if(length(make_float2(rc1.b.y,rc1.b.z)) > S.D/2)
    {
        /*p=0;
        P[index*7+p++]=P1;
        P[index*7+p++]=P2;
        P[index*7+p++]=P3;
        P[index*7+p++]=P5;
        P[index*7+p++]=nan3;
        P[index*7+p++]=nan3;
        P[index*7+p++]=nan3;*/
        //P[index*7]=make_float3(400,400,400);//error4
        return;
    }

    ns = normalize(rc1.b);

    float3 v5 = findAlpha( -ns, v4,1,S.m2 );

    //For intensity calculation
    float P10 = acos(dot(-ns, v4));

    float X = S.l1 + 2*S.g + S.ll;
    t = ( X - P5.x ) / v5.x;

    float3 P6 = P5 + v5*t;

    float3 v6 = findAlpha( n, v5,2,S.m2 );

    //For intensity calculation
    float P11 = acos(dot(n, v5));

    t = (S.lCCD - P6.x ) / v6.x;

    float3 P7 = P6 + v6*t;

    /*p=0;
    P[index*7+p++]=P1;
    P[index*7+p++]=P2;
    P[index*7+p++]=P3;
    P[index*7+p++]=P4;
    P[index*7+p++]=P5;
    P[index*7+p++]=P6;
    P[index*7+p++]=P7;*/

    /*if(IM==NULL || IM==0)
    {
        //P[index*7]=make_float3(500,500,500);//error5
        return;//no need to calculate image
    }*/

    /*float dist=length(P1-P2)+length(P2-P3)+
                length(P3-P4)+length(P4-P5)+
                length(P5-P6)+length(P6-P7);
    float3 vR = normalize(P7-P6);
    float alp = acos(dot(make_float3(1,0,0),vR));*/
    float W  = S.shX + ( S.CCDW/2.0f +P7.y)/S.PixSize;
    float Hi = S.shY + ( S.CCDH/2.0f +P7.z)/S.PixSize;
    //float value=cos(alp)/(dist*dist);

    //Recording position of rays and a number of rays that walk into the cell
    float value=1.0f;
    float* val0;
    val0=PX+(unsigned int)round(Hi)*4+(unsigned int)round(W)*480;
    atomicAdd(val0, P2.x);
    val0=PX+1+(unsigned int)round(Hi)*4+(unsigned int)round(W)*480;
    atomicAdd(val0, P2.y);
    val0=PX+2+(unsigned int)round(Hi)*4+(unsigned int)round(W)*480;
    atomicAdd(val0, P2.z);
    val0=PX+3+(unsigned int)round(Hi)*4+(unsigned int)round(W)*480;
    atomicAdd(val0, value);//+1

    //The calculation of energy loss,  caused by reflection on lens surfaces and rising distance
    value=0.01f*(length(P1-P2) + length(P2-P3));
    value*=value;//fast square
    float Ka1 = cos(P8)/value;
    value=0.01f*length(P3-P4);
    value*=value;
    float Ka2 = Ka1*cos(P9)/value;
    value=0.01f*length(P4-P5);
    value*=value;
    float Ka3 = Ka2*cos(P10)/value;
    value=cos(P11);//in calculation intensive code calculating same cosine twice isn't wise
    value*=value;
    float Ka4 = Ka3*value;
    value=0.01f*(length(P5-P6) + length(P6-P7));
    Ka4/=value;
    value=1.0f/Ka4;
    val0=IC+(unsigned int)round(Hi)+(unsigned int)round(W)*480;
    atomicAdd(val0, value);

    //float* val0=IM+(unsigned int)round(Hi)+(unsigned int)round(W)*480;
    //atomicAdd(val0, value);
}
}//extern "C"
