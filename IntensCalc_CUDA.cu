#include "hip/hip_runtime.h"
/** \file IntensCalc_CUDA.cu
 * \author Tomasz Jakubczyk
 * \brief plik z implementacjami funkcji wywołujących CUDA'ę
 *
 *
 *
 */

#include "mex.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_vector_types.h"
#include "IntensCalc_CUDA_kernel.cuh"
#include "MovingAverage_CUDA_kernel.cuh"

#ifdef DEBUG
extern unsigned short* previewFa;/**< klatka po obliczeniu wartości pixeli */
unsigned short* previewFa=nullptr;

extern short* previewFb;/**< czerwona klatka po demosaicu */
short* previewFb=nullptr;

extern float* previewFc;/**< czerwona klatka po nałożeniu obrazu korekcyjnego */
float* previewFc=nullptr;

extern float* previewFd;/**< czerwona klatka po sumowaniu pixeli */
float* previewFd=nullptr;
#endif // DEBUG

__host__
//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

__host__
/** \brief compute grid and thread block size for a given number of elements
 *
 * \param n uint
 * \param blockSize uint
 * \param numBlocks uint&
 * \param numThreads uint&
 * \return void
 *
 */
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

hipError_t err;
char* dev_buff=NULL;
unsigned short* dev_frame=NULL;
short* dev_outArray=NULL;

int* dev_ipR=NULL;
int ipR_Size=0;
int* dev_ipG=NULL;
int ipG_Size=0;
int* dev_ipB=NULL;
int ipB_Size=0;
float* dev_ICR_N=NULL;
float* dev_ICG_N=NULL;
float* dev_ICB_N=NULL;
int* dev_I_S_R=NULL;
int* dev_I_S_G=NULL;
int* dev_I_S_B=NULL;
float* dev_IR=NULL;
float* dev_IG=NULL;
float* dev_IB=NULL;
float* dev_sIR=NULL;
float* dev_sIG=NULL;
float* dev_sIB=NULL;
float* dev_RR=NULL;
float* dev_RG=NULL;
float* dev_RB=NULL;
unsigned char* dev_BgMask=NULL;
float* dev_BgValue=NULL;
float BgMask_Size=0;
float lastProbablyCorrectBgValue=60;

int licznik_klatek=0;
short previewFb2[640*480];

extern "C"
{

void setupCUDA_IC()
{
    /**< przygotowanie CUDA'y */

    checkCudaErrors(hipSetDevice(0));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipSetDevice): %s\n", hipGetErrorString(err));
    }

    checkCudaErrors(hipFree(0));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipFree(0)): %s\n", hipGetErrorString(err));
    }

    checkCudaErrors(hipMalloc((void**)&dev_buff, sizeof(char)*640*480*2));
    checkCudaErrors(hipMalloc((void**)&dev_frame, sizeof(unsigned short)*640*480));
    checkCudaErrors(hipMalloc((void**)&dev_outArray, sizeof(short)*640*480*3));
    checkCudaErrors(hipMalloc((void**)&dev_BgValue, sizeof(float)));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMemset(dev_buff,0,sizeof(char)*640*480*2));
    checkCudaErrors(hipMemset(dev_frame,0,sizeof(unsigned short)*640*480));
    checkCudaErrors(hipMemset(dev_outArray,0,sizeof(short)*640*480*3));
    checkCudaErrors(hipMemset(dev_BgValue,0,sizeof(float)));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipMemset): %s\n", hipGetErrorString(err));
    }
}

void setMasksAndImagesAndSortedIndexes(
    int* ipR,int ipR_size,int* ipG,int ipG_size,int* ipB, int ipB_size,
    float* ICR_N, float* ICG_N, float* ICB_N,
    int* I_S_R, int* I_S_G, int* I_S_B,
    unsigned char* BgMask, float BgMaskSize)
{
    ipR_Size=ipR_size;
    ipG_Size=ipG_size;
    ipB_Size=ipB_size;

    checkCudaErrors(hipSetDevice(0));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipSetDevice): %s\n", hipGetErrorString(err));
    }

    checkCudaErrors(hipMalloc((void**)&dev_ipR, sizeof(int)*ipR_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_ipG, sizeof(int)*ipG_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_ipB, sizeof(int)*ipB_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_ICR_N, sizeof(float)*ipR_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_ICG_N, sizeof(float)*ipG_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_ICB_N, sizeof(float)*ipB_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_I_S_R, sizeof(int)*ipR_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_I_S_G, sizeof(int)*ipG_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_I_S_B, sizeof(int)*ipB_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }

    checkCudaErrors(hipMalloc((void**)&dev_IR, sizeof(float)*ipR_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_IG, sizeof(float)*ipG_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_IB, sizeof(float)*ipB_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }

    checkCudaErrors(hipMalloc((void**)&dev_sIR, sizeof(float)*ipR_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_sIG, sizeof(float)*ipG_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_sIB, sizeof(float)*ipB_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }

    checkCudaErrors(hipMalloc((void**)&dev_RR, sizeof(float)*700));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_RG, sizeof(float)*700));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_RB, sizeof(float)*700));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    /** \todo pobrać zmienne z wymiarami maski tła
     */
    checkCudaErrors(hipMalloc((void**)&dev_BgMask, sizeof(unsigned char)*640*480));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }

    checkCudaErrors(hipMemcpy((void*)dev_ipR, ipR, sizeof(int)*ipR_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_ipG, ipG, sizeof(int)*ipG_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_ipB, ipB, sizeof(int)*ipB_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_ICR_N, ICR_N, sizeof(float)*ipR_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_ICG_N, ICG_N, sizeof(float)*ipG_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    //return;
    checkCudaErrors(hipMemcpy((void*)dev_ICB_N, ICB_N, sizeof(float)*ipB_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    //return;
    checkCudaErrors(hipMemcpy((void*)dev_I_S_R, I_S_R, sizeof(int)*ipR_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_I_S_G, I_S_G, sizeof(int)*ipG_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_I_S_B, I_S_B, sizeof(int)*ipB_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_BgMask, BgMask, sizeof(unsigned char)*640*480, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }

    BgMask_Size=BgMaskSize;
}

void copyBuff(char* buff)
{
    /**< kopiujemy na kartę */
    checkCudaErrors(hipSetDevice(0));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipSetDevice): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMemcpy((void*)dev_buff, buff, sizeof(char)*640*480*2, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
    }
}

//extern unsigned short previewFa[640*480];


void doIC(float* I_Red, float* I_Green, float* I_Blue)
{
    uint numThreads, numBlocks;
    computeGridSize(640*480, 512, numBlocks, numThreads);
    unsigned int dimGridX=numBlocks<65535?numBlocks:65535;
    unsigned int dimGridY=numBlocks/65535+1;
    dim3 dimGrid(dimGridX,dimGridY);

    /**< Jeśli tutaj będzie działało za wolno, to można wykozystać dodatkowy wątek CPU i CUDA streams */
    aviGetValueD<<< dimGrid, numThreads >>>(dev_buff,dev_frame,640*480);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(aviGetValueD): %s\n", hipGetErrorString(err));
    }
    #ifdef DEBUG
    if(licznik_klatek==1)
    checkCudaErrors(hipMemcpy((void*)previewFa,dev_frame,sizeof(unsigned short)*640*480,hipMemcpyDeviceToHost));
    #endif // DEBUG

    /**< demosaic */
    demosaicD<<< dimGrid, numThreads >>>(dev_frame,640*480,dev_outArray);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(demosaicD): %s\n", hipGetErrorString(err));
    }

    #ifdef DEBUG
    if(licznik_klatek==1)
    checkCudaErrors(hipMemcpy((void*)previewFb,dev_outArray+640*480*2,sizeof(short)*640*480,hipMemcpyDeviceToHost));
    #endif // DEBUG

    if(ipR_Size>0)
    {
        if(licznik_klatek++<20)/**< debug */
        {
            printf("frame: %d\n",licznik_klatek);
            checkCudaErrors(hipMemcpy((void*)previewFb2,dev_outArray+640*480*2,sizeof(short)*640*480,hipMemcpyDeviceToHost));
            for(int i=0;i<480;i++)//480
            {
                for(int j=0;j<640;j++)//640
                {
                    if(i%16==8 && j%16==8)
                    printf("%d ",previewFb2[i*640+j]>=1000?1:0);
                }
                if(i%16==8)
                printf("\n");
            }
            printf("\n");
        }
        /**< obliczyć wartość tła */
        computeGridSize(640*480, 512, numBlocks, numThreads);
        unsigned int dimGridX=numBlocks<65535?numBlocks:65535;
        unsigned int dimGridY=numBlocks/65535+1;
        dim3 dimGrid0(dimGridX,dimGridY);
        checkCudaErrors(hipMemset(dev_BgValue,0,sizeof(float)));
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(hipMemset): %s\n", hipGetErrorString(err));
        }
        getBgD<<< dimGrid0, numThreads >>>(dev_outArray+640*480*2,dev_BgMask,dev_BgValue);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(getBgD R): %s\n", hipGetErrorString(err));
        }
        //dev_BgValue[0]=(float)dev_BgValue[0]/(float)dev_BgMaskSize[0];
        float tmpBgValue=0.0f;
        checkCudaErrors(hipMemcpy((void*)&tmpBgValue,dev_BgValue,sizeof(float),hipMemcpyDeviceToHost));
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(hipMemcpyDeviceToHost): %s\n", hipGetErrorString(err));
        }
        tmpBgValue/=BgMask_Size;
        if(tmpBgValue>=200.0f)
        {
            //printf("tmpBgValue: %f, ",tmpBgValue);
            tmpBgValue=lastProbablyCorrectBgValue;
        }
        else
        {
            lastProbablyCorrectBgValue+=tmpBgValue;
            lastProbablyCorrectBgValue/=2.0f;
        }
        checkCudaErrors(hipMemset(dev_BgValue,tmpBgValue,sizeof(float)));
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(hipMemset): %s\n", hipGetErrorString(err));
        }

        /**< nałożyć maskę i skorygować */
        computeGridSize(ipR_Size, 512, numBlocks, numThreads);
        dimGridX=numBlocks<65535?numBlocks:65535;
        dimGridY=numBlocks/65535+1;
        dim3 dimGrid(dimGridX,dimGridY);
        correctionD<<< dimGrid, numThreads >>>(dev_outArray+640*480*2,dev_ipR,ipR_Size,dev_ICR_N,dev_IR,dev_BgValue);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(correctionD R): %s\n", hipGetErrorString(err));
        }

        #ifdef DEBUG
        checkCudaErrors(hipMemcpy((void*)previewFc,dev_IR,sizeof(float)*ipR_Size,hipMemcpyDeviceToHost));
        #endif // DEBUG
        /**< przydatna sztuczka do podglądania w matlabie:
        tmpIM=zeros(640,480,'single');
        tmpIM=reshape(tmpIM,640*480,[]);
        tmpIM(ipR)=prevRC;
        tmpIM=reshape(tmpIM,640,480);
        imtool(tmpIM')
         */

        /**< średnia krocząca */
        MovingAverageD<<< dimGrid, numThreads >>>(dev_IR,ipR_Size,dev_I_S_R,dev_sIR,64.0f);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(MovingAverageD): %s\n", hipGetErrorString(err));
        }

        #ifdef DEBUG
        checkCudaErrors(hipMemcpy((void*)previewFd,dev_sIR,sizeof(float)*ipR_Size,hipMemcpyDeviceToHost));
        #endif // DEBUG

        DivD<<< dimGrid, numThreads >>>(ipR_Size,dev_sIR,64.0f);

        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(DivD): %s\n", hipGetErrorString(err));
        }

        /**< wybór reprezentantów */
        computeGridSize(700, 512, numBlocks, numThreads);
        dimGridX=numBlocks<65535?numBlocks:65535;
        dimGridY=numBlocks/65535+1;
        dim3 dimGrid2(dimGridX,dimGridY);

        chooseRepresentativesD<<< dimGrid2, numThreads >>>(dev_sIR,ipR_Size,dev_RR,700);

        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(chooseRepresentativesD): %s\n", hipGetErrorString(err));
        }
    }
    if(ipG_Size>0)
    {
        computeGridSize(ipG_Size, 512, numBlocks, numThreads);
        unsigned int dimGridX=numBlocks<65535?numBlocks:65535;
        unsigned int dimGridY=numBlocks/65535+1;
        dim3 dimGrid(dimGridX,dimGridY);
        correctionD<<< dimGrid, numThreads >>>(dev_outArray+640*480,dev_ipG,ipG_Size,dev_ICG_N,dev_IG,dev_BgValue);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(correctionD G): %s\n", hipGetErrorString(err));
        }

        MovingAverageD<<< dimGrid, numThreads >>>(dev_IG,ipG_Size,dev_I_S_G,dev_sIG,64.0f);

        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(MovingAverageD): %s\n", hipGetErrorString(err));
        }

        DivD<<< dimGrid, numThreads >>>(ipG_Size,dev_sIG,64.0f);

        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(DivD): %s\n", hipGetErrorString(err));
        }

        computeGridSize(700, 512, numBlocks, numThreads);
        dimGridX=numBlocks<65535?numBlocks:65535;
        dimGridY=numBlocks/65535+1;
        dim3 dimGrid2(dimGridX,dimGridY);

        chooseRepresentativesD<<< dimGrid2, numThreads >>>(dev_sIG,ipG_Size,dev_RG,700);

        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(chooseRepresentativesD): %s\n", hipGetErrorString(err));
        }
    }
    if(ipB_Size>0)
    {
        computeGridSize(ipB_Size, 512, numBlocks, numThreads);
        unsigned int dimGridX=numBlocks<65535?numBlocks:65535;
        unsigned int dimGridY=numBlocks/65535+1;
        dim3 dimGrid(dimGridX,dimGridY);
        correctionD<<< dimGrid, numThreads >>>(dev_outArray,dev_ipB,ipB_Size,dev_ICB_N,dev_IB,dev_BgValue);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(correctionD B): %s\n", hipGetErrorString(err));
        }

        MovingAverageD<<< dimGrid, numThreads >>>(dev_IB,ipB_Size,dev_I_S_B,dev_sIB,64.0f);

        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(MovingAverageD): %s\n", hipGetErrorString(err));
        }

        DivD<<< dimGrid, numThreads >>>(ipB_Size,dev_sIB,64.0f);

        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(DivD): %s\n", hipGetErrorString(err));
        }

        computeGridSize(700, 512, numBlocks, numThreads);
        dimGridX=numBlocks<65535?numBlocks:65535;
        dimGridY=numBlocks/65535+1;
        dim3 dimGrid2(dimGridX,dimGridY);

        chooseRepresentativesD<<< dimGrid2, numThreads >>>(dev_sIB,ipB_Size,dev_RB,700);

        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(chooseRepresentativesD): %s\n", hipGetErrorString(err));
        }
    }

    checkCudaErrors(hipMemcpy((void*)I_Red,dev_RR,sizeof(float)*700,hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy((void*)I_Green,dev_RG,sizeof(float)*700,hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy((void*)I_Blue,dev_RB,sizeof(float)*700,hipMemcpyDeviceToHost));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipMemcpyDeviceToHost): %s\n", hipGetErrorString(err));
    }
}

void freeCUDA_IC()
{
    checkCudaErrors(hipFree(dev_buff));
    checkCudaErrors(hipFree(dev_frame));
    checkCudaErrors(hipFree(dev_outArray));

    checkCudaErrors(hipFree(dev_ipR));
    checkCudaErrors(hipFree(dev_ipG));
    checkCudaErrors(hipFree(dev_ipB));
    checkCudaErrors(hipFree(dev_ICR_N));
    checkCudaErrors(hipFree(dev_ICG_N));
    checkCudaErrors(hipFree(dev_ICB_N));
    checkCudaErrors(hipFree(dev_I_S_R));
    checkCudaErrors(hipFree(dev_I_S_G));
    checkCudaErrors(hipFree(dev_I_S_B));
    checkCudaErrors(hipFree(dev_IR));
    checkCudaErrors(hipFree(dev_IG));
    checkCudaErrors(hipFree(dev_IB));
    checkCudaErrors(hipFree(dev_sIR));
    checkCudaErrors(hipFree(dev_sIG));
    checkCudaErrors(hipFree(dev_sIB));
    checkCudaErrors(hipFree(dev_RR));
    checkCudaErrors(hipFree(dev_RG));
    checkCudaErrors(hipFree(dev_RB));
    checkCudaErrors(hipFree(dev_BgMask));
    checkCudaErrors(hipFree(dev_BgValue));

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipFree): %s\n", hipGetErrorString(err));
    }
    hipProfilerStop();
}

}
