#include "hip/hip_runtime.h"
/** \file IntensCalc_CUDA.cu
 * \author Tomasz Jakubczyk
 * \brief plik z implementacjami funkcji wywołujących CUDA'ę
 *
 *
 *
 */

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_vector_types.h"
#include "IntensCalc_CUDA_kernel.cuh"
#include "MovingAverage_CUDA_kernel.cuh"

__host__
//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

__host__
/** \brief compute grid and thread block size for a given number of elements
 *
 * \param n uint
 * \param blockSize uint
 * \param numBlocks uint&
 * \param numThreads uint&
 * \return void
 *
 */
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

hipError_t err;
char* dev_buff=NULL;
unsigned short* dev_frame=NULL;
short* dev_outArray=NULL;

int* dev_ipR=NULL;
int ipR_Size=0;
int* dev_ipG=NULL;
int ipG_Size=0;
int* dev_ipB=NULL;
int ipB_Size=0;
float* dev_ICR_N=NULL;
float* dev_ICG_N=NULL;
float* dev_ICB_N=NULL;
int* dev_I_S_R=NULL;
int* dev_I_S_G=NULL;
int* dev_I_S_B=NULL;
float* dev_IR=NULL;
float* dev_IG=NULL;
float* dev_IB=NULL;

extern "C"
{

void setupCUDA_IC()
{
    /**< przygotowanie CUDA'y */

    checkCudaErrors(hipSetDevice(0));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipSetDevice): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_buff, sizeof(char)*640*480*2));
    checkCudaErrors(hipMalloc((void**)&dev_frame, sizeof(unsigned short)*640*480));
    checkCudaErrors(hipMalloc((void**)&dev_outArray, sizeof(short)*640*480*3));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMemset(dev_buff,0,sizeof(char)*640*480*2));
    checkCudaErrors(hipMemset(dev_frame,0,sizeof(unsigned short)*640*480));
    checkCudaErrors(hipMemset(dev_outArray,0,sizeof(short)*640*480*3));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipMemset): %s\n", hipGetErrorString(err));
    }
}

void setMasksAndImagesAndSortedIndexes(
    int* ipR,int ipR_size,int* ipG,int ipG_size,int* ipB, int ipB_size,
    float* ICR_N, float* ICG_N, float* ICB_N,
    int* I_S_R, int* I_S_G, int* I_S_B)
{
    ipR_Size=ipR_size;
    ipG_Size=ipG_size;
    ipB_Size=ipB_size;

    checkCudaErrors(hipSetDevice(0));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipSetDevice): %s\n", hipGetErrorString(err));
    }

    checkCudaErrors(hipMalloc((void**)&dev_ipR, sizeof(int)*ipR_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_ipG, sizeof(int)*ipG_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_ipB, sizeof(int)*ipB_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_ICR_N, sizeof(float)*ipR_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_ICG_N, sizeof(float)*ipG_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_ICB_N, sizeof(float)*ipB_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_I_S_R, sizeof(int)*ipR_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_I_S_G, sizeof(int)*ipG_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_I_S_B, sizeof(int)*ipB_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }

    checkCudaErrors(hipMalloc((void**)&dev_IR, sizeof(float)*ipR_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_IG, sizeof(float)*ipG_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_IB, sizeof(float)*ipB_size));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }

    checkCudaErrors(hipMemcpy((void*)dev_ipR, ipR, sizeof(int)*ipR_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_ipG, ipG, sizeof(int)*ipG_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_ipB, ipB, sizeof(int)*ipB_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_ICR_N, ICR_N, sizeof(float)*ipR_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_ICG_N, ICG_N, sizeof(float)*ipG_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    //return;
    checkCudaErrors(hipMemcpy((void*)dev_ICB_N, ICB_N, sizeof(float)*ipB_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    //return;
    checkCudaErrors(hipMemcpy((void*)dev_I_S_R, I_S_R, sizeof(int)*ipR_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_I_S_G, I_S_G, sizeof(int)*ipG_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
    checkCudaErrors(hipMemcpy((void*)dev_I_S_B, I_S_B, sizeof(int)*ipB_size, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
        return;
    }
}

void copyBuff(char* buff)
{
    /**< kopiujemy na kartę */
    checkCudaErrors(hipSetDevice(0));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipSetDevice): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMemcpy((void*)dev_buff, buff, sizeof(char)*640*480*2, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
    }
}

void doIC(float* I_Red, float* I_Green, float* I_Blue)
{
    uint numThreads, numBlocks;
    computeGridSize(640*480, 512, numBlocks, numThreads);
    unsigned int dimGridX=numBlocks<65535?numBlocks:65535;
    unsigned int dimGridY=numBlocks/65535+1;
    dim3 dimGrid(dimGridX,dimGridY);

    /**< Jeśli tutaj będzie działało za wolno, to można wykozystać dodatkowy wątek CPU i CUDA streams */
    aviGetValueD<<< dimGrid, numThreads >>>(dev_buff,dev_frame,640*480);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(aviGetValueD): %s\n", hipGetErrorString(err));
    }

    /**< demosaic */
    demosaicD<<< dimGrid, numThreads >>>(dev_frame,640*480,dev_outArray);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(demosaicD): %s\n", hipGetErrorString(err));
    }

    /**< nałożyć maskę i skorygować */
    if(ipR_Size>0)
    {
        computeGridSize(ipR_Size, 512, numBlocks, numThreads);
        unsigned int dimGridX=numBlocks<65535?numBlocks:65535;
        unsigned int dimGridY=numBlocks/65535+1;
        dim3 dimGrid(dimGridX,dimGridY);
        correctionD<<< dimGrid, numThreads >>>(dev_outArray,dev_ipR,ipR_Size,dev_ICR_N,dev_IR);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(correctionD R): %s\n", hipGetErrorString(err));
        }
    }
    if(ipG_Size>0)
    {
        computeGridSize(ipG_Size, 512, numBlocks, numThreads);
        unsigned int dimGridX=numBlocks<65535?numBlocks:65535;
        unsigned int dimGridY=numBlocks/65535+1;
        dim3 dimGrid(dimGridX,dimGridY);
        correctionD<<< dimGrid, numThreads >>>(dev_outArray+640*480,dev_ipG,ipG_Size,dev_ICG_N,dev_IG);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(correctionD G): %s\n", hipGetErrorString(err));
        }
    }
    if(ipB_Size>0)
    {
        computeGridSize(ipB_Size, 512, numBlocks, numThreads);
        unsigned int dimGridX=numBlocks<65535?numBlocks:65535;
        unsigned int dimGridY=numBlocks/65535+1;
        dim3 dimGrid(dimGridX,dimGridY);
        correctionD<<< dimGrid, numThreads >>>(dev_outArray+640*480*2,dev_ipB,ipB_Size,dev_ICB_N,dev_IB);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("hipError_t(correctionD B): %s\n", hipGetErrorString(err));
        }
    }

    unsigned short int klatka[307200];
    checkCudaErrors(hipMemcpy((void*)klatka,dev_frame,sizeof(unsigned short)*640*480,hipMemcpyDeviceToHost));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipMemcpyDeviceToHost): %s\n", hipGetErrorString(err));
    }
}

void freeCUDA_IC()
{
    checkCudaErrors(hipFree(dev_buff));
    checkCudaErrors(hipFree(dev_frame));
    checkCudaErrors(hipFree(dev_outArray));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipFree): %s\n", hipGetErrorString(err));
    }
    hipProfilerStop();
}

}
