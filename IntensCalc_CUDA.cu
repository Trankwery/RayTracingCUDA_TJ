#include "hip/hip_runtime.h"
/** \file IntensCalc_CUDA.cu
 * \author Tomasz Jakubczyk
 * \brief plik z implementacjami funkcji wywołujących CUDA'ę
 *
 *
 *
 */

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "hip/hip_vector_types.h"
#include "IntensCalc_CUDA_kernel.cuh"

__host__
//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

__host__
/** \brief compute grid and thread block size for a given number of elements
 *
 * \param n uint
 * \param blockSize uint
 * \param numBlocks uint&
 * \param numThreads uint&
 * \return void
 *
 */
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

hipError_t err;
char* dev_buff=NULL;
unsigned short* dev_frame=NULL;

extern "C"
{

void setupCUDA_IC()
{
    /**< przygotowanie CUDA'y */

    checkCudaErrors(hipSetDevice(0));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipSetDevice): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMalloc((void**)&dev_buff, sizeof(char)*640*480*2));
    checkCudaErrors(hipMalloc((void**)&dev_frame, sizeof(unsigned short)*640*480));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Malloc): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMemset(dev_buff,0,sizeof(char)*640*480*2));
    checkCudaErrors(hipMemset(dev_frame,0,sizeof(unsigned short)*640*480));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipMemset): %s\n", hipGetErrorString(err));
    }
}

void copyBuff(char* buff)
{
    /**< kopiujemy na kartę */
    checkCudaErrors(hipSetDevice(0));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipSetDevice): %s\n", hipGetErrorString(err));
    }
    checkCudaErrors(hipMemcpy((void*)dev_buff, buff, sizeof(char)*640*480*2, hipMemcpyHostToDevice));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(Memcpy): %s\n", hipGetErrorString(err));
    }
}

void doIC(float* I_Red, float* I_Green, float* I_Blue)
{
    uint numThreads, numBlocks;
    computeGridSize(640*480, 512, numBlocks, numThreads);
    unsigned int dimGridX=numBlocks<65535?numBlocks:65535;
    unsigned int dimGridY=numBlocks/65535+1;
    dim3 dimGrid(dimGridX,dimGridY);

    /**< Jeśli tutaj będzie działało za wolno, to można wykozystać dodatkowy wątek CPU i CUDA streams */
    aviGetValueD<<< dimGrid, numThreads >>>(dev_buff,dev_frame,640*480);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(aviGetValueD): %s\n", hipGetErrorString(err));
    }
    unsigned short int klatka[307200];
    checkCudaErrors(hipMemcpy((void*)klatka,dev_frame,sizeof(unsigned short)*640*480,hipMemcpyDeviceToHost));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipMemcpyDeviceToHost): %s\n", hipGetErrorString(err));
    }
    /*for(int i=0;i<480;i++)
    {
        for(int j=0;j<640;j++)
        {
            printf("%d ",klatka[i*640+j]);
        }
        printf("\n");
    }*/
}

void freeCUDA_IC()
{
    checkCudaErrors(hipFree(dev_buff));
    checkCudaErrors(hipFree(dev_frame));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("hipError_t(hipFree): %s\n", hipGetErrorString(err));
    }
}

}
