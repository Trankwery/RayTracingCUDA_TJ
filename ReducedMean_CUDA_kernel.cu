#include "hip/hip_runtime.h"
/** \file ReducedMean_CUDA_kernel.cu
 * \author Tomasz Jakubczyk
 * \brief kernel function
 *
 *
 *
 */
#define WIN32
#include<stdlib.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "hip/hip_vector_types.h"
#include "hip/hip_math_constants.h"

extern "C"
{

__global__
void ReducedMeanD(float* Theta_S,unsigned int Theta_S_size, float deltaT, unsigned int max_nom, float* I, float* I_S, float* nTheta, float* nI, float* counter)
{
    // unique block index inside a 3D block grid
    const unsigned int blockId = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    uint index = __mul24(blockId,blockDim.x) + threadIdx.x;
    if(index>=Theta_S_size)
        return;
    float value;
    float* val0;

    unsigned int nom=floor((Theta_S[index]-Theta_S[0])/deltaT);
    if(nom>max_nom || nom<=0)
        return;
    //if(((Theta_S[0]+deltaT*nom)<=Theta_S[index]) && ((Theta_S[0]+deltaT*(nom+1))>=Theta_S[index]))
    //{
        val0=nTheta+nom-1;
        value=Theta_S[index];
        atomicAdd(val0, value);
        val0=nI+nom-1;
        value=I[(unsigned int)round(I_S[index])];
        atomicAdd(val0, value);
        val0=counter+nom-1;
        value=1.0f;
        atomicAdd(val0, value);
    //}
}

}

