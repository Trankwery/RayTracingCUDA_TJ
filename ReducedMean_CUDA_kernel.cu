#include "hip/hip_runtime.h"
/** \file ReducedMean_CUDA_kernel.cu
 * \author Tomasz Jakubczyk
 * \brief kernel function
 *
 *
 *
 */
#define WIN32
#include<stdlib.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "hip/hip_vector_types.h"
#include "hip/hip_math_constants.h"

extern "C"
{

__global__
void ReducedMeanD(float* Theta_S, float deltaT, unsigned int max_nom, float* I, float* I_S, float* nTheta, float* nI, float* counter)
{
    // unique block index inside a 3D block grid
    const unsigned int blockId = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    uint index = __mul24(blockId,blockDim.x) + threadIdx.x;
    float value;
    float* val0;
    #pragma unroll
    for(unsigned int nom=1;nom<=max_nom)
    {
        if(((Theta_S[0]+deltaT*nom)<=Theta_S[index]) && ((Theta_S[0]+deltaT*(nom+1))<=Theta_S[index]))
        {
            val0=nTheta+nom-1;
            value=Theta_S[index];
            atomicAdd(val0, value);
            val0=nI+nom-1;
            value=I[(unsigned int)I_S[index]];
            atomicAdd(val0, value);
            val0=counter+nom-1;
            value=1.0f;
            atomicAdd(val0, value);
        }
    }
}

}

