#include "hip/hip_runtime.h"
/** \file IntensCalc_CUDA_kernel.cu
 * \author Tomasz Jakubczyk
 * \brief funkcje CUDA na GPU
 *
 *
 *
 */


#define WIN32
#include<stdlib.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "hip/hip_vector_types.h"
#include "hip/hip_math_constants.h"
#include "JunkStruct.h"

#include <stdio.h>

extern "C"
{


/** \brief każdy watek sprawdza swój bajt danych i jeśli jest to początek jednego z kodów
 * JUNK albo header to zapisuje je synchronizowanej do listy
 * \param DataSpace char* wskaźnik na dane
 * \param junkList JunkStruct* wskaźnik na listę junk
 * \param junkCounter long int* wskaźnik na licznik znalezionych sekcji junk
 * \param headerList long int* wskaźnik na listę nagłówków
 * \param headerCounter long int* wskaźnik na licznik znalezionych nagłówków
 * \return void
 *
 */
__global__
void findJunkAndHeadersD(char* DataSpace,JunkStruct* junkList,long int* junkCounter,long int* headerList,long int* headerCounter)
{
    // unique block index inside a 3D block grid
    const unsigned int blockId = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    uint index = __mul24(blockId,blockDim.x) + threadIdx.x;
    if(index>=655350*2-8)
        return;
    const char frameStartCode[8]={'0','0','d','b',0x00,0x60,0x09,0x00};
    const char frameStartCodeS[8]={'0','0','d','c',0x00,0x60,0x09,0x00};
    const char junkCode[]="JUNK";
    bool junkB=true;
    for(int i=0;i<4;i++)
    {
        junkB&=junkCode[i]==DataSpace[index+i];
        if(!junkB)
        {
            break;
        }
    }
    if(junkB)
    {
        long int tmpJunkCounter=atomicAdd(junkCounter,1);
        junkList[tmpJunkCounter].position=index;
        junkList[tmpJunkCounter].size=*(long int*)(DataSpace+index+4);
        return;
    }
    bool headerB=true;
    for(int i=0;i<header.size;i++)
    {
        headerB&=frameStartCode[i]==DataSpace[index+i] || frameStartCodeS[i]==DataSpace[index+i];
        if(!headerB)
        {
            return;
        }
    }
    if(headerB)
    {
        long int tmpHeaderCounter=atomicAdd(headerCounter,1);
        headerList[tmpHeaderCounter]=index;
    }
}


/** \brief wyliczanie wartości pixeli z bajtów filmu
 *
 * \param buff char*
 * \param frame unsigned short*
 * \param frame_size unsigned int
 * \return void
 *
 */
__global__
void aviGetValueD(char* buff, unsigned short* frame, unsigned int frame_size)
{
    // unique block index inside a 3D block grid
    const unsigned int blockId = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    uint index = __mul24(blockId,blockDim.x) + threadIdx.x;
    if(index>=frame_size)
        return;
    const unsigned char reverse6bitLookupTable[]={
0x00,0x20,0x10,0x30,0x08,0x28,0x18,0x38,0x04,0x24,0x14,0x34,0x0C,0x2C,0x1C,0x3C,
0x02,0x22,0x12,0x32,0x0A,0x2A,0x1A,0x3A,0x06,0x26,0x16,0x36,0x0E,0x2E,0x1E,0x3E,
0x01,0x21,0x11,0x31,0x09,0x29,0x19,0x39,0x05,0x25,0x15,0x35,0x0D,0x2D,0x1D,0x3D,
0x03,0x23,0x13,0x33,0x0B,0x2B,0x1B,0x3B,0x07,0x27,0x17,0x37,0x0F,0x2F,0x1F,0x3F};
/**< tablica odwracająca kolejność 6 młodszych bitów */

    unsigned short bl,bh;
    bh=0x00FF&buff[2*index];/**< CUDA zdaje się sama nie zerować starszego bajtu ze śmieci */
    bh=bh<<6;
    bl=0x00FF&buff[2*index+1];
    bl=bl>>2;
    bl=0x00FF&(unsigned short)reverse6bitLookupTable[(unsigned char)bl];
    frame[index]=bh|bl;
}

#define I frame
/** \brief demosaic GRBG bruteforce!
 *
 * \param frame unsigned short*
 * \param frame_size unsigned int
 * \param outArray short*
 * \return void
 *
 */
__global__
void demosaicD(unsigned short* frame, unsigned int frame_size, short* outArray)
{
    // unique block index inside a 3D block grid
    const unsigned int blockId = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    uint index = __mul24(blockId,blockDim.x) + threadIdx.x;
    if(index>=frame_size)
        return;

    int j=index/640;
    int i=index%640;
    int wid=640, len=480;
    int x_max=wid-1, y_max=len-1;
    int im1=0,ip1=0,jm1=0,jp1=0;
    int lenxwid=len*wid;

    jm1=j==0?j+1:j-1;//j-1
    jp1=j==y_max?j-1:j+1;//j+1

    im1=i==0?i+1:i-1;//i-1
    ip1=i==x_max?i-1:i+1;//i+1
    if((i&1)==0)
    {
        if((j&1)==0)//R(G)R
        {
            outArray[i+j*wid]=(I[i+jm1*wid]+I[i+jp1*wid])>>1;//B
        }
        else//G(B)G
        {
            outArray[i+j*wid]=I[i+wid*j];//B
        }
    }
    else
    {
        if((j&1)==0)//G(R)G
        {
            outArray[i+j*wid]=(I[im1+wid*jm1]+I[ip1+wid*jp1]+I[im1+wid*jp1]+I[ip1+wid*jm1])>>2;//B
        }
        else//B(G)B
        {
            outArray[i+j*wid]=(I[im1+wid*j]+I[ip1+wid*j])>>1;//B
        }
    }

    jm1=j==0?j+1:j-1;//j-1
    jp1=j==y_max?j-1:j+1;//j+1

    im1=i==0?i+1:i-1;//i-1
    ip1=i==x_max?i-1:i+1;//i+1
    if((i&1)==0)
    {
        if((j&1)==0)//R(G)R
        {
            outArray[i+j*wid+lenxwid]=I[i+j*wid];//G
        }
        else//G(B)G
        {
            outArray[i+j*wid+lenxwid]=(I[im1+wid*j]+I[ip1+wid*j]+I[i+wid*jm1]+I[i+wid*jp1])>>2;//G
        }
    }
    else
    {
        if((j&1)==0)//G(R)G
        {
            outArray[i+j*wid+lenxwid]=(I[im1+wid*j]+I[ip1+wid*j]+I[i+wid*jm1]+I[i+wid*jp1])>>2;//G
        }
        else//B(G)B
        {
            outArray[i+j*wid+lenxwid]=I[i+wid*j];//G
        }
    }

    jm1=j==0?j+1:j-1;//j-1
    jp1=j==y_max?j-1:j+1;//j+1

    im1=i==0?i+1:i-1;//i-1
    ip1=i==x_max?i-1:i+1;//i+1
    if((i&1)==0)
    {
        if((j&1)==0)//R(G)R
        {
            outArray[i+j*wid+2*lenxwid]=(I[im1+j*wid]+I[ip1+j*wid])>>1;//R
        }
        else//G(B)G
        {
            outArray[i+j*wid+2*lenxwid]=(I[im1+jm1*wid]+I[ip1+wid*jp1]+I[im1+wid*jp1]+I[ip1+wid*jm1])>>2;//R
        }
    }
    else
    {
        if((j&1)==0)//G(R)G
        {
            outArray[i+j*wid+2*lenxwid]=I[i+wid*j];//R
        }
        else//B(G)B
        {
            outArray[i+j*wid+2*lenxwid]=(I[i+wid*jm1]+I[i+wid*jp1])>>1;//R
        }
    }
}

/** \brief oblicza wartość tła
 *
 * \param color short* klatka w wybranym kolorze
 * \param BgMask unsigned char* maska tła
 * \param BgValue float* zwracana wartość tła
 * \return void
 *
 */
__global__
void getBgD(short* color, unsigned char* BgMask, float* BgValue)
{
    // unique block index inside a 3D block grid
    const unsigned int blockId = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    uint index = __mul24(blockId,blockDim.x) + threadIdx.x;
    if(index>=640*480)
        return;
    int j=index/640;
    int i=index%640;
    int wid=640, len=480;
    if(BgMask[j+i*len]==0)
        return;
    float value=(float)color[i+j*wid];//j+i*len
    if(i<wid/2)
    {
        atomicAdd(BgValue,value);
    }
    else
    {
        float* tmpBg=&(BgValue[1]);
        atomicAdd(tmpBg,value);
    }
}

/** \brief nałożenie maski na kolor klatki i podzielenie przez obraz korekcyjny
 *
 * \param color short* klatka w wybranym kolorze
 * \param mask int* nakładana maska
 * \param mask_size int rozmiar maski
 * \param IC float* obraz korekcyjny
 * \param I float* zwracana skorygowana klatka w wybranym kolorze
 * \return void
 *
 */
__global__
void correctionD(short* color, int* mask, int mask_size, float* IC, float* I, float* BgValue)
{
    // unique block index inside a 3D block grid
    const unsigned int blockId = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    uint index = __mul24(blockId,blockDim.x) + threadIdx.x;
    if(index>=mask_size)
        return;
    int maskIndex=mask[index]-1;
    #ifndef ONE_BACKGROUND
    float tmpBgValue=(maskIndex%640<320)?BgValue[0]:BgValue[1];
    #else
    float tmpBgValue=(BgValue[0]+BgValue[1])*0.5f;
    #endif // ONE_BACKGROUND
    float tmpColor=(float)color[maskIndex]-tmpBgValue;
    if(tmpColor<=0)
    {
        I[index]=0;
    }
    else
    {
        I[index]=tmpColor/IC[index];
    }
}

/** \brief wybiera równomiernie rozłożone punkty
 *
 * \param I float* duży zbiór danych
 * \param I_size int rozmiar dużego zbioru
 * \param R float* zbiór wybranych danych
 * \param R_size int rozmiar wybranych danych
 * \return void
 *
 */
__global__
void chooseRepresentativesD(float* I, int I_size, float* R, int R_size)
{
    // unique block index inside a 3D block grid
    const unsigned int blockId = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    uint index = __mul24(blockId,blockDim.x) + threadIdx.x;
    if(index>=R_size)
        return;
    R[index]=I[index*(I_size-1)/R_size+1];
}

}
