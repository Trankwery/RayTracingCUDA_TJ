#include "hip/hip_runtime.h"
/** \file MovingAverage_CUDA_kernel.cu
 * \author Tomasz Jakubczyk
 * \brief plik z kernelem CUDA wyg�adzaj�cym za pomoc� �redniej krocz�cej
 *
 *
 *
 */

#define WIN32
#include<stdlib.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "hip/hip_vector_types.h"
#include "hip/hip_math_constants.h"

#define STEP 64

extern "C"
{

__global__
void MovingAverageD(float* I, unsigned int I_size, int* I_S, float* sI, float step)
{
    // unique block index inside a 3D block grid
    const unsigned int blockId = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    uint index = __mul24(blockId,blockDim.x) + threadIdx.x;
    if(index>=I_size)
        return;
    float value;
    float* val0;

    #pragma unroll
    for(unsigned int i=0;i<step && index+i<I_size;i++)
    {
        val0=sI+index+i;
        value=I[(unsigned int)round(I_S[index]-1.0f)];
        atomicAdd(val0, value);
    }
}

__global__
void DivD(unsigned int I_size, float* sI, float step)
{
    // unique block index inside a 3D block grid
    const unsigned int blockId = blockIdx.x //1D
        + blockIdx.y * gridDim.x //2D
        + gridDim.x * gridDim.y * blockIdx.z; //3D
    uint index = __mul24(blockId,blockDim.x) + threadIdx.x;
    if(index>=I_size)
        return;
    if(index>=step-1)
    {
        sI[index]/=(float)step;
    }
    else
    {
        sI[index]/=(float)(index+1);
    }
}

}
