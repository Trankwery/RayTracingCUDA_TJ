#include "hip/hip_runtime.h"
/** \file RayTraceCUDA.cu
 * \author Tomasz Jakubczyk
 * \brief Implementation of RayTrace function
 * which calls RayTraceD CUDA kernels
 */
#define WIN32
#include "HandlesStructures.cuh"
#include "RayTraceCUDA_kernel.cuh"
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "hip/hip_vector_types.h"
#include <stdlib.h>
#include<stdio.h>
#include "mex.h"

extern "C"
{
    __host__
    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    __host__
    /** \brief compute grid and thread block size for a given number of elements
     *
     * \param n uint
     * \param blockSize uint
     * \param numBlocks uint&
     * \param numThreads uint&
     * \return void
     *
     */
    void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }

    __host__
    /** \brief calculate ray tracing.
     * call RayTraceD CUDA kernels
     * \param Br float*
     * \param Vb int*
     * \param VH float*
     * \param Vb_length int
     * \param VH_length int
     * \param S HandlesStructures
     * \param IM float3* image. must be zeroed before ray tracing.
     * to skip image calculation set pointer to NULL
     * \param P float3*
     * \return void
     *
     */
    void RayTrace(float* Br, int Br_size, float* Vb, float* VH, int Vb_length, int VH_length, HandlesStructures S, float* IC, int IC_size, float* PX)
    {
        hipError_t err;
        float* dev_Br=0;
        float* dev_Vb=0;
        float* dev_VH=0;
        float* dev_IC=0;
        float* dev_PX=0;
        checkCudaErrors(hipMalloc((void**)&dev_Br, sizeof(float)*Br_size));
        err = hipGetLastError();
        if (err != hipSuccess)
		{
			printf("hipError_t(hipMalloc((void**)&dev_Br, sizeof(float)*Br_size)): %s\n", hipGetErrorString(err));
		}
        checkCudaErrors(hipMemcpy((void*)dev_Br, Br, sizeof(float)*Br_size, hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void**)&dev_Vb, sizeof(float)*Vb_length));
        checkCudaErrors(hipMemcpy((void*)dev_Vb, Vb, sizeof(float)*Vb_length, hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void**)&dev_VH, sizeof(float)*VH_length));
        checkCudaErrors(hipMemcpy((void*)dev_VH, VH, sizeof(float)*VH_length, hipMemcpyHostToDevice));
		checkCudaErrors(hipMalloc((void**)&dev_IC, sizeof(float)*IC_size));
		checkCudaErrors(hipMemset(dev_IC,0,sizeof(float)*IC_size));
        checkCudaErrors(hipMalloc((void**)&dev_PX, sizeof(float)*4*480*640));
        checkCudaErrors(hipMemset(dev_PX,0,sizeof(float)*4*480*640));

        uint numThreads, numBlocks;
        computeGridSize(VH_length*Vb_length, 512, numBlocks, numThreads);
        unsigned int dimGridX=numBlocks<65535?numBlocks:65535;
        unsigned int dimGridY=numBlocks/65535+1;
        dim3 dimGrid(dimGridX,dimGridY);

        err = hipGetLastError();
        if (err != hipSuccess)
		{
			printf("1cudaError(while GPU memory allocation): %s\n", hipGetErrorString(err));
		}

        //system("pause");
        printf("dev_IC:%d\n",dev_IC);

        printf("numBlocks: %d\n",numBlocks);
        printf("numThreads: %d\n",numThreads);
        printf("dimGrid.x: %d\n",dimGrid.x);
        printf("dimGrid.y: %d\n",dimGrid.y);

        RayTraceD<<< dimGrid, numThreads >>>(dev_Br,dev_Vb,dev_VH,Vb_length,VH_length,S,dev_IC,dev_PX);
        //RayTraceD<<< 2, 25 >>>(dev_Br,dev_Vb,dev_VH,Vb_length,VH_length,S,dev_IM,dev_P);

        err = hipGetLastError();
        if (err != hipSuccess)
		{
			printf("2cudaError(while CUDA kernel execution): %s\n", hipGetErrorString(err));
		}

        checkCudaErrors(hipMemcpy((void*)PX,dev_PX,sizeof(float)*4*480*640,hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy((void*)IC,dev_IC,sizeof(float)*IC_size,hipMemcpyDeviceToHost));

		err = hipGetLastError();
        if (err != hipSuccess)
		{
			printf("3cudaError(while hipMemcpy): %s\n", hipGetErrorString(err));
		}

		checkCudaErrors(hipFree(dev_IC));

        checkCudaErrors(hipFree(dev_PX));
        checkCudaErrors(hipFree(dev_VH));
        checkCudaErrors(hipFree(dev_Vb));
        checkCudaErrors(hipFree(dev_Br));
    }
}
